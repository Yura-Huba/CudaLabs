﻿
#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"

#define _USE_MATH_DEFINES
#define ARRAY_SIZE 1000000000

#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include <stdlib.h>
 

typedef double type;

__global__ void calculateSin(type* ptrToArray, size_t arraySize) {
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < arraySize) {
		ptrToArray[index] = sinf(((type)(index % 360)) * M_PI / 180);
	}
}

double calculateSineError(type* ptrToArray, size_t arraySize) {
	double totalError = 0;
	for (int i = 0; i < arraySize; i++) {
		totalError += abs(sin((i % 360) * M_PI / 180) - ptrToArray[i]);
	}
	return totalError / arraySize;
}

int main() {
	const size_t arraySize = ARRAY_SIZE;
	type* ptrToArrayOnDevice;
	hipError_t hipError_t;

	int deviceCount = 0;
	hipError_t = hipGetDeviceCount(&deviceCount);

	if (hipSuccess != hipError_t) {
		printf("Description: %s\n", hipGetErrorString(hipError_t));
		return EXIT_FAILURE;
	}

	int device = 0;
	hipSetDevice(device);
	
	hipDeviceProp_t deviceProp;
	hipError_t = hipGetDeviceProperties(&deviceProp, device);

	if (hipSuccess != hipError_t) {
		printf("Description: %s\n", hipGetErrorString(hipError_t));
		return EXIT_FAILURE;
	}

	hipError_t = hipMalloc(&ptrToArrayOnDevice, sizeof(type) * arraySize);

	if (hipSuccess != hipError_t) {
		printf("Description: %s\n", hipGetErrorString(hipError_t));
		return EXIT_FAILURE;
	}

	dim3 threads = dim3(deviceProp.maxThreadsPerBlock);
	dim3 blocks = dim3((unsigned int) ceil((double) arraySize / deviceProp.maxThreadsPerBlock));

	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	calculateSin <<<blocks, threads >>> (ptrToArrayOnDevice, arraySize);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time = %0.8f\n", elapsedTime);


	type* ptrToArrayOnHost;
	ptrToArrayOnHost = (type*)malloc(sizeof(type) * arraySize);

	if (ptrToArrayOnHost == NULL) {
		printf("No memory host\n");
		return EXIT_FAILURE;
	}

	hipError_t = hipMemcpy(ptrToArrayOnHost, ptrToArrayOnDevice, sizeof(type) * arraySize, hipMemcpyDeviceToHost);

	if (hipSuccess != hipError_t) {
		printf("Description: %s\n", hipGetErrorString(hipError_t));
		return EXIT_FAILURE;
	}

	printf("Error = %0.10f \n", calculateSineError(ptrToArrayOnHost, arraySize));

	hipFree(ptrToArrayOnDevice);
	free(ptrToArrayOnHost);

	return EXIT_SUCCESS;
}